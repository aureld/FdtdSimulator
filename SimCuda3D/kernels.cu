#include "hip/hip_runtime.h"
// kernels.cu : update equations for E and H - Cuda enabled
// Aurelien Duval 2015
//see ACES J. 25(4) p303 (2010)

#include <stdio.h>
#include "hip/hip_runtime.h"
#include "SimCuda3D/cuda_macros.h"
#include "SimCuda3D/cuda_protos.h"
#include "SimCuda3D/Cuda_grid.h"
#include <math.h>

//array indexing macros
#define IDX(i, j, k) ((i) + (j) * (g->nx) + (k) * (g->nx) * (g->ny) )
#define K(index) (index / (g->nx * g->ny))
#define J(index) ((index - (K(index)*g->nx*g->ny))/g->nx)
#define I(index) ((index) - J(index) * g->nx - K(index) * g->nx * g->ny)



//initialize all device field arrays to zero before use (Seems there's no 
__global__ void Cuda_initFieldArrays( grid *g)
{
    //grid stride loop
    for (int index = blockIdx.x * blockDim.x + threadIdx.x; index < g->domainSize; index += blockDim.x * gridDim.x)
    {
        g->ex[index] = 0.0;  
        g->ey[index] = 0.0; 
        g->ez[index] = 0.0; 
        g->hx[index] = 0.0; 
        g->hy[index] = 0.0; 
        g->hz[index] = 0.0; 
        g->Ca[index] = 0.0;
        g->Cb1[index] = 0.0;
        g->Cb2[index] = 0.0;
        g->Db1[index] = 0.0;
        g->Db2[index] = 0.0;
    }
}


//update equations for H fields - Cuda naive approach
__device__ void Cuda_updateH(grid *g)
{
    int i, j, k, pos;

    //grid stride loop
    for (int index = blockIdx.x * blockDim.x + threadIdx.x; index < g->domainSize; index += blockDim.x * gridDim.x)
    {
        i = I(index);
        j = J(index);
        k = K(index);
        pos = IDX(i, j, k);

        if (i > g->nx-1  || j > g->ny-1 || k > g->nz-1) return;
        if (i <= 1 || j <= 1 || k <= 1) return;

        Cuda_updateHComponent(0, g->hx, i, j, k, pos, g);
        Cuda_updateHComponent(1, g->hy, i, j, k, pos, g);
        Cuda_updateHComponent(2, g->hz, i, j, k, pos, g);

    }

    return;
}

__device__ inline void Cuda_updateHComponent(int component, float *h, int i, int j, int k, int pos, grid *g)
{
    float e1a, e1b, e2a, e2b;
    switch (component)
    {
    case 0: //X
        e1a = g->ey[pos];
        e1b = g->ey[IDX(i, j, k - 1)];
        e2a = g->ez[pos];
        e2b = g->ez[IDX(i, j - 1, k)];
        break;
    case 1: //Y
        e1a = g->ez[pos];
        e1b = g->ez[IDX(i - 1, j, k)];
        e2a = g->ex[pos];
        e2b = g->ex[IDX(i, j, k - 1)];
        break;
    case 2: //Z
        e1a = g->ex[pos];
        e1b = g->ex[IDX(i, j - 1, k)];
        e2a = g->ey[pos];
        e2b = g->ey[IDX(i - 1, j, k)];
        break;
    }
    h[pos] = h[pos] + g->Db1[pos] * (e1a - e1b) - g->Db2[pos] * (e2a - e2b);
}


__device__ void Cuda_updateHBoundaries(grid *g)
{
    int i, j, k, pos;
    //grid stride loop
    for (int index = blockIdx.x * blockDim.x + threadIdx.x; index < g->domainSize; index += blockDim.x * gridDim.x)
    {
        i = I(index);
        j = J(index);
        k = K(index);
        pos = IDX(i, j, k);

        if (i == 0)             //left
            Cuda_updateHBoundaryMinusX(i, j, k, pos, g);
        if (i == g->nx - 1)     //right
            Cuda_updateHBoundaryPlusX(i, j, k, pos, g);
        if (j == 0)             //bottom
            Cuda_updateHBoundaryMinusY(i, j, k, pos, g);
        if (j == g->ny - 1)     //top
            Cuda_updateHBoundaryPlusY(i, j, k, pos, g);
        if (k == 0)             //back
            Cuda_updateHBoundaryMinusZ(i, j, k, pos, g);
        if (k == g->nz - 1)     //front
            Cuda_updateHBoundaryPlusZ(i, j, k, pos, g);

    }

}

//update equations for E fields - Cuda naive approach
__device__ void Cuda_updateE(grid *g)
{
    int i, j, k, pos;

    //grid stride loop
    for (int index = blockIdx.x * blockDim.x + threadIdx.x; index < g->domainSize; index += blockDim.x * gridDim.x)
    {
        i = I(index);
        j = J(index);
        k = K(index);
        pos = IDX(i, j, k);
        if (i > g->nx - 1 || j > g->ny - 1 || k > g->nz - 1) return;

        Cuda_updateEComponent(0, g->ex, i, j, k, pos, g);
        Cuda_updateEComponent(1, g->ey, i, j, k, pos, g);
        Cuda_updateEComponent(2, g->ez, i, j, k, pos, g);
    }
    return;
}

__device__ inline void Cuda_updateEComponent( int component, float *e, int i, int j, int k, int pos, grid *g)
{
    float h1a, h1b, h2a, h2b;
    switch (component)
    {
    case 0:
        h1a = g->hz[IDX(i, j + 1, k)];
        h1b = g->hz[pos];
        h2a = g->hy[IDX(i, j, k + 1)];
        h2b = g->hy[pos];
        break;
    case 1:
        h1a = g->hx[IDX(i, j, k + 1)];
        h1b = g->hx[pos];
        h2a = g->hz[IDX(i + 1, j, k)];
        h2b = g->hz[pos];
        break;
    case 2:
        h1a = g->hy[IDX(i + 1, j, k)];
        h1b = g->hy[pos];
        h2a = g->hx[IDX(i, j + 1, k)];
        h2b = g->hx[pos];
        break;
    }
    e[pos] = g->Ca[pos] * e[pos] + g->Cb1[pos] * (h1a - h1b) - g->Cb2[pos] * (h2a - h2b);
}

__device__ void Cuda_updateEBoundaries(grid *g)
{
    int i, j, k, pos;
    //grid stride loop
    for (int index = blockIdx.x * blockDim.x + threadIdx.x; index < g->domainSize; index += blockDim.x * gridDim.x)
    {
        i = I(index);
        j = J(index);
        k = K(index);
        pos = IDX(i, j, k);

        if (i == 0)             //left
            Cuda_updateEBoundaryMinusX(i, j, k, pos, g);
        if (i == g->nx - 1)     //right
            Cuda_updateEBoundaryPlusX(i, j, k, pos, g);
        if (j == 0)             //bottom
            Cuda_updateEBoundaryMinusY(i, j, k, pos, g);
        if (j == g->ny - 1)     //top
            Cuda_updateEBoundaryPlusY(i, j, k, pos, g);
        if (k == 0)             //back
            Cuda_updateEBoundaryMinusZ(i, j, k, pos, g);
        if (k == g->nz - 1)     //front
            Cuda_updateEBoundaryPlusZ(i, j, k, pos, g);

    }

}


// update equations for E boundary fields -X side - Cuda naive approach
//PEC only for now
__device__ inline void Cuda_updateEBoundaryMinusX(int i, int j, int k, int pos, grid *g)
{
        if (j >= g->ny-1 || k >= g->nz-1) return;

        g->ex[pos] = 0.0; //PEC
        g->ey[pos] = 0.0;
        g->ez[pos] = 0.0;
}
 
// update equations for E boundary fields +X side - Cuda naive approach
//PEC only for now
__device__ inline void Cuda_updateEBoundaryPlusX(int i, int j, int k, int pos, grid *g)
{
        if (j >= g->ny-1 || k >= g->nz-1) return;

        g->ex[pos] = 0.0; //PEC
        g->ey[pos] = 0.0;
        g->ez[pos] = 0.0;
}

// update equations for E boundary fields -Y side - Cuda naive approach
//PEC only for now
__device__ inline void Cuda_updateEBoundaryMinusY(int i, int j, int k, int pos, grid *g)
{
        if (i >= g->nx-1 || k >= g->nz-1) return;

        g->ex[pos] = 0.0; //PEC
        g->ey[pos] = 0.0;
        g->ez[pos] = 0.0;
}

// update equations for E boundary fields +Y side - Cuda naive approach
//PEC only for now
__device__ inline void Cuda_updateEBoundaryPlusY(int i, int j, int k, int pos, grid *g)
{
        if (i >= g->nx-1 || k >= g->nz-1) return;

        g->ex[pos] = 0.0; //PEC
        g->ey[pos] = 0.0;
        g->ez[pos] = 0.0;
}

// update equations for E boundary fields -Z side - Cuda naive approach
//PEC only for now
__device__ inline void Cuda_updateEBoundaryMinusZ(int i, int j, int k, int pos, grid *g)
{
   if (i >= g->nx-1 || j >= g->ny-1) return;

    g->ex[pos] = 0.0; //PEC
    g->ey[pos] = 0.0;
    g->ez[pos] = 0.0;
}

// update equations for E boundary fields +Z side - Cuda naive approach
//PEC only for now
__device__ inline void Cuda_updateEBoundaryPlusZ(int i, int j, int k, int pos, grid *g)
{
    if (i >= g->nx-1 || j >= g->ny-1) return;

    g->ex[pos] = 0.0; //PEC
    g->ey[pos] = 0.0;
    g->ez[pos] = 0.0;
}


// update equations for H boundary fields -X side - Cuda naive approach
//PEC only for now
__device__ inline void Cuda_updateHBoundaryMinusX(int i, int j, int k, int pos, grid *g)
{
        if (j >= g->ny-1 || k >= g->nz-1) return;

        g->hx[pos] = 0.0; //PEC
        g->hy[pos] = 0.0;
        g->hz[pos] = 0.0;
}

// update equations for H boundary fields +X side - Cuda naive approach
//PEC only for now
__device__ inline void Cuda_updateHBoundaryPlusX(int i, int j, int k, int pos, grid *g)
{
        if (j >= g->ny-1 || k >= g->nz-1) return;

        g->hx[pos] = 0.0; //PEC
        g->hy[pos] = 0.0;
        g->hz[pos] = 0.0;
}

// update equations for H boundary fields -Y side - Cuda naive approach
//PEC only for now
__device__ inline void Cuda_updateHBoundaryMinusY(int i, int j, int k, int pos, grid *g)
{
    if (i >= g->nx-1 || k >= g->nz-1) return;

        g->hx[pos] = 0.0; //PEC
        g->hy[pos] = 0.0;
        g->hz[pos] = 0.0;
}

// update equations for H boundary fields +Y side - Cuda naive approach
//PEC only for now
__device__ inline void Cuda_updateHBoundaryPlusY(int i, int j, int k, int pos, grid *g)
{
        if (i >= g->nx-1 || k >= g->nz-1) return;

        g->hx[pos] = 0.0; //PEC
        g->hy[pos] = 0.0;
        g->hz[pos] = 0.0;
}


// update equations for H boundary fields -Z side - Cuda naive approach
//PEC only for now
__device__ inline void Cuda_updateHBoundaryMinusZ(int i, int j, int k, int pos, grid *g)
{
        if (i >= g->nx-1 || j >= g->ny-1) return;

        g->hx[pos] = 0.0; //PEC
        g->hy[pos] = 0.0;
        g->hz[pos] = 0.0;
}

// update equations for H boundary fields +Z side - Cuda naive approach
//PEC only for now
__device__ inline void Cuda_updateHBoundaryPlusZ(int i, int j, int k, int pos, grid *g)
{
      if (i >= g->nx-1 || j >= g->ny-1) return;

        g->hx[pos] = 0.0; //PEC
        g->hy[pos] = 0.0;
        g->hz[pos] = 0.0;
}




//Auxiliary source array initialization  
__global__ void Cuda_InitializeSrc(grid *g)
{
    //grid stride loop
    for (unsigned __int64 index = blockIdx.x * blockDim.x + threadIdx.x; index < g->nt; index += blockDim.x * gridDim.x)
    {
        if (index > g->nt) return;

        double d_efftime = index * g->dt;
        double envelope = 1.0 - exp(-(d_efftime / g->rTime)); //CW for now
        g->srcField[index] = g->amplitude * envelope * sin(g->omega * d_efftime + g->initPhase);
    }
    return;
}

//E field injection
//for single a point source, only 1 thread should be launched 
__device__ void Cuda_injectE(grid *g)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int i = I(index);
    int j = J(index);
    int k = K(index);
    int pos = IDX(i, j, k);

    //only 1 thread should update the field
    if (pos != g->srclinpos) return;

    float *field = NULL;
    switch (g->srcFieldComp)
    {
    case 0:
        field = g->ex;
        break;
    case 1:
        field = g->ey;
        break;
    case 2:
        field = g->ez;
        break;
    }
    field[g->srclinpos] += g->srcField[g->currentIteration];
}

//collect field data for select component and store in detector array for each timestep
__global__ void Cuda_CollectTimeSeriesData(float *component, float *field, int posx, int posy, int posz, __int64 timestep, grid *g)
{
    int pos = IDX(posx, posy, posz);
    component[timestep] = field[pos];
}


//whole step done on device
__global__ void Cuda_CalculateStep(grid *g)
{
    Cuda_updateE(g);
    Cuda_injectE(g);
    Cuda_updateEBoundaries(g);
    Cuda_updateH(g);
    Cuda_updateHBoundaries(g);

}
